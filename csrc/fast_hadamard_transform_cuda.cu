#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// #pragma once

#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#include "fast_hadamard_transform.h"
#include "fast_hadamard_transform_common.h"
#include "fast_hadamard_transform_special.h"
#include "static_switch.h"

template<int kNThreads_, int kLogN_, typename input_t_>
struct fast_hadamard_transform_kernel_traits {
    using input_t = input_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kLogN = kLogN_;
    static constexpr int N = 1 << kLogN;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangePerVec = sizeof(float) / sizeof(input_t);
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    static constexpr int kNChunks = N / (kNElts * kNThreads);
    // We don't want to use more than 32 KB of shared memory.
    static constexpr int kSmemExchangeSize = std::min(N * 4, 32 * 1024);
    static constexpr int kNExchangeRounds = N * 4 / kSmemExchangeSize;
    static_assert(kNExchangeRounds * kSmemExchangeSize == N * 4);
    static constexpr int kSmemSize = kNThreads > 32 ? kSmemExchangeSize : 0;
};

template<int kNThreads_, int kLogN_, typename input_t_>
struct fast_hadamard_transform_12N_kernel_traits {
    using input_t = input_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kLogN = kLogN_;
    static constexpr int N = (1 << kLogN) * 12;
    static_assert(N <= 12 * 1024, "fast_hadamard_transform_12 only supports dim <= 12288");
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = 4;
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangePerVec = sizeof(float) / sizeof(input_t);
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    static constexpr int kNChunks = N / (kNElts * kNThreads);
    static_assert(kNChunks == 12);
    // We don't want to use more than 24 KB of shared memory.
    static constexpr int kSmemExchangeSize = std::min(N * 4, 24 * 1024);
    static constexpr int kNExchangeRounds = N * 4 / kSmemExchangeSize;
    static_assert(kNExchangeRounds * kSmemExchangeSize == N * 4);
    static constexpr int kSmemSize = kNThreads > 32 ? kSmemExchangeSize : 0;
};

template<int kNThreads_, int kLogN_, typename input_t_>
struct fast_hadamard_transform_20N_kernel_traits {
    using input_t = input_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kLogN = kLogN_;
    static constexpr int N = (1 << kLogN) * 20;
    static_assert(N <= 20 * 1024, "fast_hadamard_transform_20 only supports dim <= 20480");
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = 4;
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangePerVec = sizeof(float) / sizeof(input_t);
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    static constexpr int kNChunks = N / (kNElts * kNThreads);
    static_assert(kNChunks == 20);
    // We don't want to use more than 40 KB of shared memory.
    static constexpr int kSmemExchangeSize = std::min(N * 4, 40 * 1024);
    static constexpr int kNExchangeRounds = N * 4 / kSmemExchangeSize;
    static_assert(kNExchangeRounds * kSmemExchangeSize == N * 4);
    static constexpr int kSmemSize = kNThreads > 32 ? kSmemExchangeSize : 0;
};

template<int kNThreads_, int kLogN_, typename input_t_>
struct fast_hadamard_transform_28N_kernel_traits {
    using input_t = input_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kLogN = kLogN_;
    static constexpr int N = (1 << kLogN) * 28;
    static_assert(N <= 28 * 1024, "fast_hadamard_transform_28 only supports dim <= 28672");
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = 4;
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangePerVec = sizeof(float) / sizeof(input_t);
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    static constexpr int kNChunks = N / (kNElts * kNThreads);
    static_assert(kNChunks == 28);
    // We don't want to use more than 28 KB of shared memory.
    static constexpr int kSmemExchangeSize = std::min(N * 4, 28 * 1024);
    static constexpr int kNExchangeRounds = N * 4 / kSmemExchangeSize;
    static_assert(kNExchangeRounds * kSmemExchangeSize == N * 4);
    static constexpr int kSmemSize = kNThreads > 32 ? kSmemExchangeSize : 0;
};

template<int kNThreads_, int kLogN_, typename input_t_>
struct fast_hadamard_transform_40N_kernel_traits {
    using input_t = input_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kLogN = kLogN_;
    static constexpr int N = (1 << kLogN) * 40;
    static_assert(N <= 40 * 1024, "fast_hadamard_transform_40 only supports dim <= 4096");
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = 4;
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangePerVec = sizeof(float) / sizeof(input_t);
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    static constexpr int kNChunks = N / (kNElts * kNThreads);
    static_assert(kNChunks == 40);
    // We don't want to use more than 40 KB of shared memory.
    static constexpr int kSmemExchangeSize = std::min(N * 4, 40 * 1024);
    static constexpr int kNExchangeRounds = N * 4 / kSmemExchangeSize;
    static_assert(kNExchangeRounds * kSmemExchangeSize == N * 4);
    static constexpr int kSmemSize = kNThreads > 32 ? kSmemExchangeSize : 0;
};

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_12(float x[kNChunks][12]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_12(x[c]); }
}

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_20(float x[kNChunks][20]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_20(x[c]); }
}

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_28(float x[kNChunks][28]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_28(x[c]); }
}

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_40(float x[kNChunks][40]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_40(x[c]); }
}

__device__ __forceinline__ float pow2(int x) {
    return x >= 0 ? static_cast<float>(1 << x) : 1.0f / static_cast<float>(1 << (-x));
}

__device__  __forceinline__ void get_scale_and_inv(float finfo_amax, float amax, float& scale, float& scale_inv, bool round_scale) {
    if (round_scale) {
        auto exp_scale = static_cast<int>(floorf(log2f(finfo_amax / amax)));
        scale = pow2(exp_scale), scale_inv = pow2(-exp_scale);
    } else {
        scale = finfo_amax / amax, scale_inv = amax / finfo_amax;
    }
}

template<typename Ktraits, OutCastingType OutCasting=OutCastingType::out>
__global__ __launch_bounds__(std::max(Ktraits::kNThreads, 32))
void fast_hadamard_transform_kernel(HadamardParamsBase params) {
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    constexpr int kNExchangePerVec = Ktraits::kNExchangePerVec;
    constexpr int kNExchangeRounds = Ktraits::kNExchangeRounds;
    constexpr int kNChunks = Ktraits::kNChunks;
    using input_t = typename Ktraits::input_t;
    using output_t = std::conditional_t<OutCasting == OutCastingType::e4m3, uint8_t, typename Ktraits::input_t>;
    using vec_t = typename Ktraits::vec_t;

    constexpr int kLogNElts = cilog2(Ktraits::kNElts);
    static_assert(1 << kLogNElts == kNElts, "kNElts must be a power of 2");
    constexpr int kWarpSize = std::min(kNThreads, 32);
    constexpr int kLogWarpSize = cilog2(kWarpSize);
    static_assert(1 << kLogWarpSize == kWarpSize, "Warp size must be a power of 2");
    constexpr int kNWarps = kNThreads / kWarpSize;
    constexpr int kLogNWarps = cilog2(kNWarps);
    static_assert(1 << kLogNWarps == kNWarps, "kNWarps must be a power of 2");
    constexpr int kLoadsPerExchange = Ktraits::kSmemExchangeSize / (sizeof(vec_t) * kNThreads);
    static_assert(kLoadsPerExchange * sizeof(vec_t) * kNThreads == Ktraits::kSmemExchangeSize, "kSmemExchangeSize should be a power of 2");
    static_assert(kNExchangeRounds * kLoadsPerExchange * sizeof(vec_t) == kNChunks * kNElts * sizeof(float));

    constexpr int kChunksPerExchange = Ktraits::kSmemExchangeSize / (sizeof(vec_t) * kNExchangePerVec * kNThreads);
    static_assert(kChunksPerExchange * sizeof(vec_t) * kNExchangePerVec * kNThreads == Ktraits::kSmemExchangeSize);
    constexpr int kNExchanges = kNChunks / kChunksPerExchange;
    static_assert(kNExchanges * kChunksPerExchange == kNChunks);

    constexpr int num_batch_per_block = std::max(32 / kNThreads, 1);
    const int batch_id = blockIdx.x * num_batch_per_block + threadIdx.x / kNThreads;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride;
    output_t *out = reinterpret_cast<output_t *>(params.out_ptr) + batch_id * params.out_batch_stride;
    float *Scale_Inv = reinterpret_cast<float *>(params.scale_inv_ptr) + batch_id;

    float x_vals[kNChunks][kNElts];
    if (batch_id < params.batch) {
        load_input<kNChunks, kNElts, input_t, kNThreads>(x, x_vals, params.dim);
    }

    hadamard_mult_thread<kLogNElts, kNChunks>(x_vals);
    hadamard_mult_warp<kLogWarpSize, 0, kNChunks, kNElts>(x_vals);

    if constexpr (kNWarps > 1) {
        // Shared memory.
        extern __shared__ char smem_[];
        vec_t *smem_exchange = reinterpret_cast<vec_t *>(smem_);
        exchange_smem_pre<kNChunks, kChunksPerExchange, kNElts, kWarpSize, kNWarps, true, vec_t>(x_vals, smem_exchange);
        hadamard_mult_warp<kLogNWarps, 0, kNChunks, kNElts>(x_vals);
        exchange_smem_pre<kNChunks, kChunksPerExchange, kNElts, kWarpSize, kNWarps, false, vec_t>(x_vals, smem_exchange);
    }

    if constexpr (kNChunks > 1) {
        float x_vals_transposed[kNElts][kNChunks];
        #pragma unroll
        for (int c = 0; c < kNChunks; ++c) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) { x_vals_transposed[i][c] = x_vals[c][i]; }
        }
        if constexpr (kNChunks == 12) {
            hadamard_mult_thread_chunk_12<kNElts>(x_vals_transposed);
        } else if constexpr (kNChunks == 20) {
            hadamard_mult_thread_chunk_20<kNElts>(x_vals_transposed);
        } else if constexpr (kNChunks == 28) {
            hadamard_mult_thread_chunk_28<kNElts>(x_vals_transposed);
        } else if constexpr (kNChunks == 40) {
            hadamard_mult_thread_chunk_40<kNElts>(x_vals_transposed);
        } else {
            constexpr int kLogNChunks = cilog2(kNChunks);
            static_assert(1 << kLogNChunks == kNChunks, "kNChunks must be a power of 2");
            hadamard_mult_thread<kLogNChunks, kNElts>(x_vals_transposed);
        }
        #pragma unroll
        for (int c = 0; c < kNChunks; ++c) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) { x_vals[c][i] = x_vals_transposed[i][c]; }
        }
    }

    float out_scale = params.scale;
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) {
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) { x_vals[c][i] = x_vals[c][i] * out_scale; }
    }

    if (OutCasting != OutCastingType::out) {
        float amax = FP8_AMAX_MARGIN;
        // Thread amax.
        #pragma unroll
        for (int c = 0; c < kNChunks; ++c) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) { amax = std::max(amax, std::abs(x_vals[c][i])); }
        }
        // Global amax.
        #pragma unroll
        for (int lane_mask = kNThreads / 2; lane_mask > 0; lane_mask /= 2) {
            amax = fmaxf(amax, __shfl_xor_sync(0xffffffff, amax, lane_mask));
        }
        // Scale and Scale_Inv.
        float scale, scale_inv;
        get_scale_and_inv(float8e4nv_max, amax, scale, scale_inv, params.round_scale);
        if (OutCasting == OutCastingType::e4m3 and batch_id < params.batch and threadIdx.x % kNThreads == 0) {
            *Scale_Inv = scale_inv;
        }
        // Cast to e4m3.
        uint8_t x_fp8[kNChunks][kNElts];  // fp8 storage.
        #pragma unroll
        for (int c = 0; c < kNChunks; ++c) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) { x_vals[c][i] *= scale; }
            #pragma unroll
            for (int i = 0; i < kNElts; i += 2) {
                *reinterpret_cast<__hip_fp8x2_e4m3_fnuz *>(&x_fp8[c][i]) = __hip_fp8x2_e4m3_fnuz(*reinterpret_cast<float2 *>(&x_vals[c][i]));
            }
        }
        // Store outputs.
        if (batch_id < params.batch) {
            store_output<kNChunks, kNElts, output_t, kNThreads, uint8_t, OutCasting>(out, x_fp8, params.dim, scale_inv);
        }
        return;
    }

    if (batch_id < params.batch) {
        store_output<kNChunks, kNElts, output_t, kNThreads>(out, x_vals, params.dim);
    }
}

template<int kNThreads, int kLogN, typename input_t>
void fast_hadamard_transform_launch(HadamardParamsBase &params, hipStream_t stream) {
    using Ktraits = fast_hadamard_transform_kernel_traits<kNThreads, kLogN, input_t>;
    constexpr int kSmemSize = Ktraits::kSmemSize;
    constexpr int block = std::max(kNThreads, 32);
    dim3 grid((params.batch - 1) / (block / kNThreads) + 1);
    auto kernel = &fast_hadamard_transform_kernel<Ktraits>;
    if (kSmemSize >= 48 * 1024) {
        C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        }
    kernel<<<grid, block, kSmemSize, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t>
void fast_hadamard_transform_cuda(HadamardParamsBase &params, hipStream_t stream) {
    if (params.log_N == 3) {
        fast_hadamard_transform_launch<1, 3, input_t>(params, stream);
    } else if (params.log_N == 4) {
        fast_hadamard_transform_launch<2, 4, input_t>(params, stream);
    } else if (params.log_N == 5) {
        fast_hadamard_transform_launch<4, 5, input_t>(params, stream);
    } else if (params.log_N == 6) {
        fast_hadamard_transform_launch<8, 6, input_t>(params, stream);
    } else if (params.log_N == 7) {
        fast_hadamard_transform_launch<16, 7, input_t>(params, stream);
    } else if (params.log_N == 8) {
        fast_hadamard_transform_launch<32, 8, input_t>(params, stream);
    } else if (params.log_N == 9) {
        fast_hadamard_transform_launch<32, 9, input_t>(params, stream);
    } else if (params.log_N == 10) {
        fast_hadamard_transform_launch<128, 10, input_t>(params, stream);
    } else if (params.log_N == 11) {
        fast_hadamard_transform_launch<256, 11, input_t>(params, stream);
    } else if (params.log_N == 12) {
        fast_hadamard_transform_launch<256, 12, input_t>(params, stream);
    } else if (params.log_N == 13) {
        fast_hadamard_transform_launch<256, 13, input_t>(params, stream);
    } else if (params.log_N == 14) {
        fast_hadamard_transform_launch<256, 14, input_t>(params, stream);
    } else if (params.log_N == 15) {
        fast_hadamard_transform_launch<256, 15, input_t>(params, stream);
    }
}

template<int kNThreads, int kLogN, typename input_t>
void fast_hadamard_transform_12N_launch(HadamardParamsBase &params, hipStream_t stream) {
    using Ktraits = fast_hadamard_transform_12N_kernel_traits<kNThreads, kLogN, input_t>;
    constexpr int kSmemSize = Ktraits::kSmemSize;
    constexpr int block = std::max(kNThreads, 32);
    dim3 grid((params.batch - 1) / (block / kNThreads) + 1);
    OUT_CASTING_TYPE_SWITCH(params.out_casting_type, [&] {
    auto kernel = &fast_hadamard_transform_kernel<Ktraits, OutCasting>;
    if (kSmemSize >= 48 * 1024) {
        C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        }
    kernel<<<grid, block, kSmemSize, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
}

template<typename input_t>
void fast_hadamard_transform_12N_cuda(HadamardParamsBase &params, hipStream_t stream) {
    if (params.log_N == 2) {
        fast_hadamard_transform_12N_launch<1, 2, input_t>(params, stream);
    } else if (params.log_N == 3) {
        fast_hadamard_transform_12N_launch<2, 3, input_t>(params, stream);
    } else if (params.log_N == 4) {
        fast_hadamard_transform_12N_launch<4, 4, input_t>(params, stream);
    } else if (params.log_N == 5) {
        fast_hadamard_transform_12N_launch<8, 5, input_t>(params, stream);
    } else if (params.log_N == 6) {
        fast_hadamard_transform_12N_launch<16, 6, input_t>(params, stream);
    } else if (params.log_N == 7) {
        fast_hadamard_transform_12N_launch<32, 7, input_t>(params, stream);
    } else if (params.log_N == 8) {
        fast_hadamard_transform_12N_launch<64, 8, input_t>(params, stream);
    } else if (params.log_N == 9) {
        fast_hadamard_transform_12N_launch<128, 9, input_t>(params, stream);
    } else if (params.log_N == 10) {
        fast_hadamard_transform_12N_launch<256, 10, input_t>(params, stream);
    }
}

template<int kNThreads, int kLogN, typename input_t>
void fast_hadamard_transform_20N_launch(HadamardParamsBase &params, hipStream_t stream) {
    using Ktraits = fast_hadamard_transform_20N_kernel_traits<kNThreads, kLogN, input_t>;
    constexpr int kSmemSize = Ktraits::kSmemSize;
    constexpr int block = std::max(kNThreads, 32);
    dim3 grid((params.batch - 1) / (block / kNThreads) + 1);
    auto kernel = &fast_hadamard_transform_kernel<Ktraits>;
    if (kSmemSize >= 48 * 1024) {
        C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        }
    kernel<<<grid, block, kSmemSize, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t>
void fast_hadamard_transform_20N_cuda(HadamardParamsBase &params, hipStream_t stream) {
    if (params.log_N == 2) {
        fast_hadamard_transform_20N_launch<1, 2, input_t>(params, stream);
    } else if (params.log_N == 3) {
        fast_hadamard_transform_20N_launch<2, 3, input_t>(params, stream);
    } else if (params.log_N == 4) {
        fast_hadamard_transform_20N_launch<4, 4, input_t>(params, stream);
    } else if (params.log_N == 5) {
        fast_hadamard_transform_20N_launch<8, 5, input_t>(params, stream);
    } else if (params.log_N == 6) {
        fast_hadamard_transform_20N_launch<16, 6, input_t>(params, stream);
    } else if (params.log_N == 7) {
        fast_hadamard_transform_20N_launch<32, 7, input_t>(params, stream);
    } else if (params.log_N == 8) {
        fast_hadamard_transform_20N_launch<64, 8, input_t>(params, stream);
    } else if (params.log_N == 9) {
        fast_hadamard_transform_20N_launch<128, 9, input_t>(params, stream);
    } else if (params.log_N == 10) {
        fast_hadamard_transform_20N_launch<256, 10, input_t>(params, stream);
    }
}

template<int kNThreads, int kLogN, typename input_t>
void fast_hadamard_transform_28N_launch(HadamardParamsBase &params, hipStream_t stream) {
    using Ktraits = fast_hadamard_transform_28N_kernel_traits<kNThreads, kLogN, input_t>;
    constexpr int kSmemSize = Ktraits::kSmemSize;
    constexpr int block = std::max(kNThreads, 32);
    dim3 grid((params.batch - 1) / (block / kNThreads) + 1);
    auto kernel = &fast_hadamard_transform_kernel<Ktraits>;
    if (kSmemSize >= 48 * 1024) {
        C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        }
    kernel<<<grid, block, kSmemSize, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t>
void fast_hadamard_transform_28N_cuda(HadamardParamsBase &params, hipStream_t stream) {
    if (params.log_N == 2) {
        fast_hadamard_transform_28N_launch<1, 2, input_t>(params, stream);
    } else if (params.log_N == 3) {
        fast_hadamard_transform_28N_launch<2, 3, input_t>(params, stream);
    } else if (params.log_N == 4) {
        fast_hadamard_transform_28N_launch<4, 4, input_t>(params, stream);
    } else if (params.log_N == 5) {
        fast_hadamard_transform_28N_launch<8, 5, input_t>(params, stream);
    } else if (params.log_N == 6) {
        fast_hadamard_transform_28N_launch<16, 6, input_t>(params, stream);
    } else if (params.log_N == 7) {
        fast_hadamard_transform_28N_launch<32, 7, input_t>(params, stream);
    } else if (params.log_N == 8) {
        fast_hadamard_transform_28N_launch<64, 8, input_t>(params, stream);
    } else if (params.log_N == 9) {
        fast_hadamard_transform_28N_launch<128, 9, input_t>(params, stream);
    } else if (params.log_N == 10) {
        fast_hadamard_transform_28N_launch<256, 10, input_t>(params, stream);
    }
}

template<int kNThreads, int kLogN, typename input_t>
void fast_hadamard_transform_40N_launch(HadamardParamsBase &params, hipStream_t stream) {
    using Ktraits = fast_hadamard_transform_40N_kernel_traits<kNThreads, kLogN, input_t>;
    constexpr int kSmemSize = Ktraits::kSmemSize;
    constexpr int block = std::max(kNThreads, 32);
    dim3 grid((params.batch - 1) / (block / kNThreads) + 1);
    auto kernel = &fast_hadamard_transform_kernel<Ktraits>;
    if (kSmemSize >= 48 * 1024) {
        C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        }
    kernel<<<grid, block, kSmemSize, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t>
void fast_hadamard_transform_40N_cuda(HadamardParamsBase &params, hipStream_t stream) {
    if (params.log_N == 2) {
        fast_hadamard_transform_40N_launch<1, 2, input_t>(params, stream);
    } else if (params.log_N == 3) {
        fast_hadamard_transform_40N_launch<2, 3, input_t>(params, stream);
    } else if (params.log_N == 4) {
        fast_hadamard_transform_40N_launch<4, 4, input_t>(params, stream);
    } else if (params.log_N == 5) {
        fast_hadamard_transform_40N_launch<8, 5, input_t>(params, stream);
    } else if (params.log_N == 6) {
        fast_hadamard_transform_40N_launch<16, 6, input_t>(params, stream);
    } else if (params.log_N == 7) {
        fast_hadamard_transform_40N_launch<32, 7, input_t>(params, stream);
    } else if (params.log_N == 8) {
        fast_hadamard_transform_40N_launch<64, 8, input_t>(params, stream);
    } else if (params.log_N == 9) {
        fast_hadamard_transform_40N_launch<128, 9, input_t>(params, stream);
    } else if (params.log_N == 10) {
        fast_hadamard_transform_40N_launch<256, 10, input_t>(params, stream);
    }
}

template void fast_hadamard_transform_cuda<float>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_cuda<at::Half>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_cuda<at::BFloat16>(HadamardParamsBase &params, hipStream_t stream);

template void fast_hadamard_transform_12N_cuda<float>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_12N_cuda<at::Half>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_12N_cuda<at::BFloat16>(HadamardParamsBase &params, hipStream_t stream);

template void fast_hadamard_transform_20N_cuda<float>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_20N_cuda<at::Half>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_20N_cuda<at::BFloat16>(HadamardParamsBase &params, hipStream_t stream);

template void fast_hadamard_transform_28N_cuda<float>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_28N_cuda<at::Half>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_28N_cuda<at::BFloat16>(HadamardParamsBase &params, hipStream_t stream);

template void fast_hadamard_transform_40N_cuda<float>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_40N_cuda<at::Half>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_40N_cuda<at::BFloat16>(HadamardParamsBase &params, hipStream_t stream);